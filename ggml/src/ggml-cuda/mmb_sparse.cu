#include "hip/hip_runtime.h"
#include "ggml.h"
#include "common.cuh"
#include "mmb_sparse.cuh"

// GTODO: we have not tested the kernel so far, test it when batch-example could be run
template <typename T, typename type_acc, int block_size>
static __global__ void mul_mat_batch_sparse(
        const T * __restrict__ x, 
        const float * __restrict__ y, 
        const float *  __restrict__ sparse_idx, 
        const int32_t *  __restrict__ gpu_neu_idx,
        float * __restrict__ dst,

        const int64_t nrows,
        const int64_t ncols2, 
        const int64_t stride_row,  // ncols
        const int64_t src1_ncols  // token batch number
        ) {
            
    const int64_t row         = blockIdx.x;  // (0, nrows)
    const int64_t s1col_b     = blockIdx.y;   // (0, scr1_ncols) the block that responsible for the specific token in batch
    const int     tid         = threadIdx.x; // (0, 256)
    constexpr int warp_size   = ggml_cuda_get_physical_warp_size();

    int gpu_neu = gpu_neu_idx ? gpu_neu_idx[row] : row; // (one of the gpu_neurons index)

    x          += stride_row * row;
    y          += stride_row * s1col_b;
    dst        += nrows * s1col_b;
    sparse_idx += nrows * s1col_b;

    if(sparse_idx[gpu_neu] < 0.5){ // GTODO: do we need sparse_threshold?
        //printf("sparse: %d: %f\n", gpu_neu,sparse_idx[gpu_neu]);
        return;
    }

    const float2 * y2 = (const float2 *) y;

    extern __shared__ char data_mmv[];
    float * buf_iw = (float *) data_mmv;

    if (block_size > warp_size) {
        if (tid < warp_size) {
            buf_iw[tid] = 0.0f;
        }
        __syncthreads();
    }

    float sumf = 0.0f;

    if constexpr (std::is_same<T, float>::value) {
        const float2 * x2 = (const float2 *) x;

        for (int64_t col2 = tid; col2 < ncols2; col2 += block_size) {
            const float2 tmpx = x2[col2];
            const float2 tmpy = y2[col2];
            sumf += tmpx.x*tmpy.x;
            sumf += tmpx.y*tmpy.y;
        }
    } else if constexpr (std::is_same<T, half>::value) {
        const half2 * x2 = (const half2 *) x;

        if (std::is_same<type_acc, float>::value) {
            for (int64_t col2 = tid; col2 < ncols2; col2 += block_size) {
                const float2 tmpx = __half22float2(x2[col2]);
                const float2 tmpy = y2[col2];
                sumf += tmpx.x * tmpy.x;
                sumf += tmpx.y * tmpy.y;
            }
        } else {
#ifdef FP16_AVAILABLE
            half2 sumh2 = make_half2(0.0f, 0.0f);

            for (int64_t col2 = tid; col2 < ncols2; col2 += block_size) {
                const float2 tmp = y2[col2];
                sumh2 += x2[col2] * make_half2(tmp.x, tmp.y);
            }

            sumf = __low2float(sumh2) + __high2float(sumh2);
#else
            NO_DEVICE_CODE;
#endif // FP16_AVAILABLE
        }
    } else if constexpr (std::is_same<T, hip_bfloat16>::value) {
        const int * x2 = (const int *) x;
        for (int64_t col2 = tid; col2 < ncols2; col2 += block_size) {
            const int    tmpx = x2[col2];
            const float2 tmpy = y2[col2];
            sumf += float(reinterpret_cast<const hip_bfloat16 *>(&tmpx)[0]) * tmpy.x;
            sumf += float(reinterpret_cast<const hip_bfloat16 *>(&tmpx)[1]) * tmpy.y;
        }
    } else {
        static_assert(std::is_same<T, void>::value, "unsupported type");
    }

    sumf = warp_reduce_sum<warp_size>(sumf);

    if (block_size > warp_size) {
        buf_iw[tid/warp_size] = sumf;
        __syncthreads();
        if (tid >= warp_size) {
            return;
        }
        sumf = buf_iw[tid];
        sumf = warp_reduce_sum<warp_size>(sumf);
    }

    if (tid != 0) {
        return;
    }

    dst[gpu_neu] = sumf;
}

template <typename T, typename type_acc>
static void launch_mul_mat_batch_cuda_sparse(
        const T * x, const float * y, const float * sparse_idx, const int32_t * gpu_neu_idx, float * dst,
        const int64_t ncols, const int64_t nrows, const int64_t stride_row, const int64_t src1_ncols, hipStream_t stream) {
    GGML_ASSERT(ncols      % 2 == 0);
    GGML_ASSERT(stride_row % 2 == 0);

    int device;
    int warp_size;

    CUDA_CHECK(hipGetDevice(&device));
    warp_size = ggml_cuda_info().devices[device].warp_size;

    int64_t block_size_best = warp_size;
    int64_t niter_best      = (ncols + 2*warp_size - 1) / (2*warp_size);
    int64_t max_block_size  = 256;
    if(ggml_cuda_info().devices[device].cc > GGML_CUDA_CC_OFFSET_AMD && ggml_cuda_info().devices[device].cc < GGML_CUDA_CC_RDNA1) {
        max_block_size = 128;
    }
    for (int64_t block_size = 2*warp_size; block_size <= max_block_size; block_size += warp_size) {   // GTODO: we need to optimize best blocksize in batch-compute?
        const int64_t niter = (ncols + 2*block_size - 1) / (2*block_size);
        if (niter < niter_best) {
            niter_best      = niter;
            block_size_best = block_size;
        }
    }

    const int smem = warp_size*sizeof(float);
    const dim3 block_nums(nrows, src1_ncols, 1); // (neurons_num, src1_ncols, 1)
    const dim3 block_dims(block_size_best, 1, 1); // (256, 1 ,1)
    
    switch (block_size_best) { // 256
        case   32: {
            mul_mat_batch_sparse<T, type_acc,  32><<<block_nums, block_dims, smem, stream>>>
                (x, y, sparse_idx, gpu_neu_idx, dst, nrows, ncols/2, stride_row, src1_ncols);
        } break;
        case   64: {
            mul_mat_batch_sparse<T, type_acc,  64><<<block_nums, block_dims, smem, stream>>>
                (x, y, sparse_idx, gpu_neu_idx, dst, nrows, ncols/2, stride_row, src1_ncols);
        } break;
        case   96: {
            mul_mat_batch_sparse<T, type_acc,  96><<<block_nums, block_dims, smem, stream>>>
                (x, y, sparse_idx, gpu_neu_idx, dst, nrows, ncols/2, stride_row, src1_ncols);
        } break;
        case  128: {
            mul_mat_batch_sparse<T, type_acc, 128><<<block_nums, block_dims, smem, stream>>>
                (x, y, sparse_idx, gpu_neu_idx, dst, nrows, ncols/2, stride_row, src1_ncols);
        } break;
        case  160: {
            mul_mat_batch_sparse<T, type_acc, 160><<<block_nums, block_dims, smem, stream>>>
               (x, y, sparse_idx, gpu_neu_idx, dst, nrows, ncols/2, stride_row, src1_ncols);
        } break;
        case  192: {
            mul_mat_batch_sparse<T, type_acc, 192><<<block_nums, block_dims, smem, stream>>>
               (x, y, sparse_idx, gpu_neu_idx, dst, nrows, ncols/2, stride_row, src1_ncols);
        } break;
        case  224: {
            mul_mat_batch_sparse<T, type_acc, 224><<<block_nums, block_dims, smem, stream>>>
               (x, y, sparse_idx, gpu_neu_idx, dst, nrows, ncols/2, stride_row, src1_ncols);
        } break;
        case  256: {
            mul_mat_batch_sparse<T, type_acc, 256><<<block_nums, block_dims, smem, stream>>>
                (x, y, sparse_idx, gpu_neu_idx, dst, nrows, ncols/2, stride_row, src1_ncols);
        } break;
        default: {
            GGML_ABORT("fatal error");
        } break;
    }
}

template<typename T>
static void mul_mat_batch_cuda_sparse(
        const T * x, const float * y, const float * sparse_idx, const int32_t * gpu_neu_idx, float * dst,
        const int64_t ncols, const int64_t nrows, const int64_t stride_row, const int64_t src1_ncols,
        enum ggml_prec prec, hipStream_t stream) {
    if constexpr(std::is_same<T, half>::value) {
        if (prec == GGML_PREC_DEFAULT) {
            launch_mul_mat_batch_cuda_sparse<T, half>
                (x, y, sparse_idx, gpu_neu_idx, dst, ncols, nrows, stride_row, src1_ncols, stream);
            return;
        }
    }
    launch_mul_mat_batch_cuda_sparse<T, float>
        (x, y, sparse_idx, gpu_neu_idx, dst, ncols, nrows, stride_row, src1_ncols, stream);
}

void * ggml_cuda_get_tensor_data_batch(const ggml_tensor * tensor) {
    return tensor->data;
    // if (!tensor) {
    //     printf("no tensor, %s\n",tensor->name);
    //     GGML_ASSERT(false && "tensor is null");
    //     return nullptr;
    // }
    // if (!tensor->extra) {
    //     printf("no tensor-extra, %s\n",tensor->name);
    //     GGML_ASSERT(false && "tensor->extra is null");
    //     return nullptr;
    // }
    // int device = ggml_cuda_get_device();
    // auto extra = (ggml_tensor_extra_gpu *) tensor->extra;

    // if(tensor->data)
    // return extra->data_device[device];
}


void ggml_cuda_op_mul_mat_batch_sparse(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, 
    const ggml_tensor * src1, 
    ggml_tensor *       dst, 

    const char *        src0_dd_i, 
    const float *       src1_ddf_i,
    const char *        src1_ddq_i, 
    float *             dst_dd_i, 

    const int64_t       row_low, 
    const int64_t       row_high, 
    const int64_t       src1_ncols,
    const int64_t       src1_padded_row_size, 
    
    hipStream_t        stream) 
    {

    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type  == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t row_diff = row_high - row_low;

    GGML_ASSERT(ggml_cuda_get_tensor_data_batch(dst->src[2])!=nullptr  && "missing sparse_idx");

    float * sparse_idx = static_cast<float *>(ggml_cuda_get_tensor_data_batch(dst->src[2]));
    int32_t * gpu_neu_idx = dst->src[3] != NULL ? static_cast<int32_t *>(ggml_cuda_get_tensor_data_batch(dst->src[3])) : NULL;

    const int cc = ggml_cuda_info().devices[ggml_cuda_get_device()].cc;
    const enum ggml_prec prec = fast_fp16_available(cc) ? ggml_prec(dst->op_params[0]) : GGML_PREC_F32;

    const int64_t stride_row  = ne00;

    switch (src0->type) {
        case GGML_TYPE_F32: {
            const float * src0_d = (const float *) src0_dd_i;
            mul_mat_batch_cuda_sparse(src0_d, src1_ddf_i, sparse_idx, gpu_neu_idx, dst_dd_i, ne00, row_diff, stride_row, src1_ncols, prec, stream);
        } break;
        case GGML_TYPE_F16: {
            const half * src0_d = (const half *) src0_dd_i;
            mul_mat_batch_cuda_sparse(src0_d, src1_ddf_i, sparse_idx, gpu_neu_idx, dst_dd_i, ne00, row_diff, stride_row, src1_ncols, prec, stream);
        } break;
        case GGML_TYPE_BF16: {
            const hip_bfloat16 * src0_d = (const hip_bfloat16 *) src0_dd_i;
            mul_mat_batch_cuda_sparse(src0_d, src1_ddf_i, sparse_idx, gpu_neu_idx, dst_dd_i, ne00, row_diff, stride_row, src1_ncols, prec, stream);
        } break;
        default:
            GGML_ABORT("unsupported type: %s", ggml_type_name(src0->type));
    }

    GGML_UNUSED(ctx);
    GGML_UNUSED(src1);
    GGML_UNUSED(dst);
    GGML_UNUSED(src1_ddq_i);
    GGML_UNUSED(src1_ncols);
    GGML_UNUSED(src1_padded_row_size);
}
