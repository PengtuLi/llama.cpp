#include "hip/hip_runtime.h"
#include "ggml.h"
#include "common.cuh"
#include "mmv_sparse.cuh"

// the powerinfer kernel: 
static __device__ void convert_f16(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const half * x = (const half *) vx;

    // automatic half -> float type cast if dfloat == float
    v.x = x[ib + iqs + 0];
    v.y = x[ib + iqs + 1];
}

static __global__ void dequantize_mul_mat_axpy_sparse(const void * __restrict__ vx, const dfloat * __restrict__ y, float * __restrict__ dst, const int ncols, const int nrows, const int *lst, const float *idx) {
    // qk = quantized weights per x block
    // qr = number of quantized weights per data value in x block
    const int gpu_row = blockIdx.y*blockDim.y + threadIdx.y; // range from [0,nrows]
    int qk =1;
    int qr = 1;

    if (gpu_row >= nrows) {
        return;
    }
    int row = lst ? lst[gpu_row] : gpu_row;
    const int tid = threadIdx.x; // range from [0,31]
    short *d = (short *)((char *)vx + ncols * gpu_row * 2);

    if (y[row] == 0)
        return;
    if (idx[row] < 0.5f) {
        return;
    }

    extern __shared__ float shared_dst[]; // TODO:dynamic

    const int iter_stride = 2*32;
    const int vals_per_iter = iter_stride / 32; // num quantized vals per thread and i iter
    const int y_offset = qr == 1 ? 1 : qk/2;

// partial sum for each thread
    float tmp = 0.0f;
    for (int i = 0; i < ncols; i += 32) {
        shared_dst[i+tid] = 0;
    }
    __syncthreads();

    for (int i = 0; i < ncols; i += iter_stride) {
        const int col = i + vals_per_iter*tid;
        const int ib = (gpu_row*ncols + col)/qk; // x block index
        const int iqs = (col%qk)/qr; // x quant index
        const int iybs = col - col%qk; // y block start index

// processing >2 values per i iter is faster for fast GPUs
#pragma unroll
        for (int j = 0; j < vals_per_iter; j += 2) {
            // process 2 vals per j iter

            // dequantize
            // for qr = 2 the iqs needs to increase by 1 per j iter because 2 weights per data val
            dfloat2 v;
            convert_f16(vx, ib, iqs + j/qr, v);

            // matrix multiplication
            // for qr = 2 the y index needs to increase by 1 per j iter because of y_offset = qk/2
            tmp = v.x * y[row];
            shared_dst[col] = tmp;  // share_dst[col] = tmp
            tmp = v.y * y[row];
            shared_dst[col+1] = tmp; // share_dst[col+1] = tmp
            
        }
    }
    __syncthreads();

    for (int i = 0; i < ncols; i += 32) {
        atomicAdd(&dst[i+tid], shared_dst[i+tid]);
    }
}

// template <typename T, typename type_acc, int block_size>
// static __global__ void mul_mat_axpy_sparse(
//         const T * __restrict__ x, 
//         const float * __restrict__ y, 
//         const float *  __restrict__ sparse_idx, 
//         const int32_t *  __restrict__ gpu_neu_idx,
//         float * __restrict__ dst,

//         const int64_t ncols2, 
//         const int64_t stride_row
//         ) {
            
//     const int64_t row         = blockIdx.x;  // (0, nrows)
//     const int     tid         = threadIdx.x; // (0, 256)

//     int gpu_neu = gpu_neu_idx ? gpu_neu_idx[row] : row; // (one of the gpu_neurons index)
    
//     if(sparse_idx[gpu_neu] < 0.5f){ // GTODO: do we need sparse_threshold?
//         if (tid == 0) dst[gpu_neu] = 0.0f; // GTODO: this should be done in initialization. ps: outputs are different if we dont set 0 before return, meaning dst was not initialized as 0 at the beginning?
//         return;
//     }

//     constexpr int warp_size   = ggml_cuda_get_physical_warp_size();

//     x += row*stride_row;

//     const float2 * y2 = (const float2 *) y;

//     extern __shared__ char data_mmv[];
//     float * buf_iw = (float *) data_mmv;

//     if (block_size > warp_size) {
//         if (tid < warp_size) {
//             buf_iw[tid] = 0.0f;
//         }
//         __syncthreads();
//     }

//     float sumf = 0.0f;

//     if constexpr (std::is_same<T, float>::value) {
//         const float2 * x2 = (const float2 *) x;

//         for (int64_t col2 = tid; col2 < ncols2; col2 += block_size) {
//             const float2 tmpx = x2[col2];
//             const float2 tmpy = y2[col2];
//             sumf += tmpx.x*tmpy.x;
//             sumf += tmpx.y*tmpy.y;
//         }
//     } else if constexpr (std::is_same<T, half>::value) {
//         const half2 * x2 = (const half2 *) x;

//         if (std::is_same<type_acc, float>::value) {
//             for (int64_t col2 = tid; col2 < ncols2; col2 += block_size) {
//                 const float2 tmpx = __half22float2(x2[col2]);
//                 const float2 tmpy = y2[col2];
//                 sumf += tmpx.x * tmpy.x;
//                 sumf += tmpx.y * tmpy.y;
//             }
//         } else {
// #ifdef FP16_AVAILABLE
//             half2 sumh2 = make_half2(0.0f, 0.0f);

//             for (int64_t col2 = tid; col2 < ncols2; col2 += block_size) {
//                 const float2 tmp = y2[col2];
//                 sumh2 += x2[col2] * make_half2(tmp.x, tmp.y);
//             }

//             sumf = __low2float(sumh2) + __high2float(sumh2);
// #else
//             NO_DEVICE_CODE;
// #endif // FP16_AVAILABLE
//         }
//     } else if constexpr (std::is_same<T, hip_bfloat16>::value) {
//         const int * x2 = (const int *) x;
//         for (int64_t col2 = tid; col2 < ncols2; col2 += block_size) {
//             const int    tmpx = x2[col2];
//             const float2 tmpy = y2[col2];
//             sumf += float(reinterpret_cast<const hip_bfloat16 *>(&tmpx)[0]) * tmpy.x;
//             sumf += float(reinterpret_cast<const hip_bfloat16 *>(&tmpx)[1]) * tmpy.y;
//         }
//     } else {
//         static_assert(std::is_same<T, void>::value, "unsupported type");
//     }

//     sumf = warp_reduce_sum<warp_size>(sumf);

//     if (block_size > warp_size) {
//         buf_iw[tid/warp_size] = sumf;
//         __syncthreads();
//         if (tid >= warp_size) {
//             return;
//         }
//         sumf = buf_iw[tid];
//         sumf = warp_reduce_sum<warp_size>(sumf);
//     }

//     if (tid != 0) {
//         return;
//     }

//     dst[gpu_neu] = sumf;
// }

template <typename T, typename type_acc>
static void launch_mul_mat_axpy_cuda_sparse(
        const T * x, const float * y, const float * sparse_idx, const int32_t * gpu_neu_idx, float * dst,
        const int64_t ncols, const int64_t nrows, const int64_t src_ncols, hipStream_t stream) {
    
    // vec_axpy
    if(src_ncols == 1){
        // the lanucher for powerinfer kernel: 
        const dim3 block_nums(1, nrows, 1);
        const dim3 block_dims(32, 1, 1);

        dequantize_mul_mat_axpy_sparse<<<block_nums, block_dims, ncols*sizeof(float), stream>>>(x, y, dst, ncols, nrows, gpu_neu_idx, sparse_idx);
        
        // GGML_ASSERT(ncols      % 2 == 0);
        // GGML_ASSERT(stride_row % 2 == 0);

        // int device;
        // int warp_size;

        // CUDA_CHECK(hipGetDevice(&device));
        // warp_size = ggml_cuda_info().devices[device].warp_size;

        // int64_t block_size_best = warp_size;
        // int64_t niter_best      = (ncols + 2*warp_size - 1) / (2*warp_size);
        // int64_t max_block_size  = 256;
        // if(ggml_cuda_info().devices[device].cc > GGML_CUDA_CC_OFFSET_AMD && ggml_cuda_info().devices[device].cc < GGML_CUDA_CC_RDNA1) {
        //     max_block_size = 128;
        // }

        // // GTODO: understand why we choose block_size like this, do we need to change this in sparse inference?
        // for (int64_t block_size = 2*warp_size; block_size <= max_block_size; block_size += warp_size) {
        //     const int64_t niter = (ncols + 2*block_size - 1) / (2*block_size);
        //     if (niter < niter_best) {
        //         niter_best      = niter;
        //         block_size_best = block_size;
        //     }
        // }

        // const int smem = warp_size*sizeof(float);
        // const dim3 block_nums(nrows, 1, 1); // (neurons_num, 1, 1)
        // const dim3 block_dims(block_size_best, 1, 1); // (256, 1 ,1)
        
        // switch (block_size_best) { // 256
        //     case   32: {
        //         mul_mat_axpy_sparse<T, type_acc,  32><<<block_nums, block_dims, smem, stream>>>
        //             (x, y, sparse_idx, gpu_neu_idx, dst, ncols/2, stride_row);
        //     } break;
        //     case   64: {
        //         mul_mat_axpy_sparse<T, type_acc,  64><<<block_nums, block_dims, smem, stream>>>
        //             (x, y, sparse_idx, gpu_neu_idx, dst, ncols/2, stride_row);
        //     } break;
        //     case   96: {
        //         mul_mat_axpy_sparse<T, type_acc,  96><<<block_nums, block_dims, smem, stream>>>
        //             (x, y, sparse_idx, gpu_neu_idx, dst, ncols/2, stride_row);
        //     } break;
        //     case  128: {
        //         mul_mat_axpy_sparse<T, type_acc, 128><<<block_nums, block_dims, smem, stream>>>
        //             (x, y, sparse_idx, gpu_neu_idx, dst, ncols/2, stride_row);
        //     } break;
        //     case  160: {
        //         mul_mat_axpy_sparse<T, type_acc, 160><<<block_nums, block_dims, smem, stream>>>
        //            (x, y, sparse_idx, gpu_neu_idx, dst, ncols/2, stride_row);
        //     } break;
        //     case  192: {
        //         mul_mat_axpy_sparse<T, type_acc, 192><<<block_nums, block_dims, smem, stream>>>
        //            (x, y, sparse_idx, gpu_neu_idx, dst, ncols/2, stride_row);
        //     } break;
        //     case  224: {
        //         mul_mat_axpy_sparse<T, type_acc, 224><<<block_nums, block_dims, smem, stream>>>
        //            (x, y, sparse_idx, gpu_neu_idx, dst, ncols/2, stride_row);
        //     } break;
        //     case  256: {
        //         mul_mat_axpy_sparse<T, type_acc, 256><<<block_nums, block_dims, smem, stream>>>
        //             (x, y, sparse_idx, gpu_neu_idx, dst, ncols/2, stride_row);
        //     } break;
        //     default: {
        //         GGML_ABORT("fatal error");
        //     } break;
        // }
    }
    else{ // batch_axpy
        GGML_ASSERT(false && "GTODO: batch axpy need to be done");
    }

}

template<typename T>
static void mul_mat_axpy_cuda_sparse(
        const T * x, const float * y, const float * sparse_idx, const int32_t * gpu_neu_idx, float * dst,
        const int64_t ncols, const int64_t nrows, const int64_t src_ncols,
        enum ggml_prec prec, hipStream_t stream) {
    if constexpr(std::is_same<T, half>::value) {
        if (prec == GGML_PREC_DEFAULT) {
            launch_mul_mat_axpy_cuda_sparse<T, half>
                (x, y, sparse_idx, gpu_neu_idx, dst, ncols, nrows, src_ncols, stream);
            return;
        }
    }
    launch_mul_mat_axpy_cuda_sparse<T, float>
        (x, y, sparse_idx, gpu_neu_idx, dst, ncols, nrows, src_ncols, stream);
}

// GTODO: this is very hacky, we need to add more safety check later
// but more importantly, what's the diffence between tensor->data & tensor-extra->data_device[device]? which to load???
void * ggml_cuda_get_tensor_data_axpy(const ggml_tensor * tensor) {
    return tensor->data;
    // if (!tensor) {
    //     printf("no tensor, %s\n",tensor->name);
    //     GGML_ASSERT(false && "tensor is null");
    //     return nullptr;
    // }
    // if (!tensor->extra) {
    //     printf("no tensor-extra, %s\n",tensor->name); 
    //     GGML_ASSERT(false && "tensor->extra is null"); sparse_idx在这里会报错, saprse_idx is only at tensor->data 
    //     return nullptr;
    // }
    // int device = ggml_cuda_get_device();
    // auto extra = (ggml_tensor_extra_gpu *) tensor->extra;

    // if(tensor->data)
    // return extra->data_device[device];
}


void ggml_cuda_op_axpy_sparse(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, 
    const ggml_tensor * src1, 
    ggml_tensor *       dst, 

    const char *        src0_dd_i, 
    const float *       src1_ddf_i,
    const char *        src1_ddq_i, 
    float *             dst_dd_i, 

    const int64_t       row_low, 
    const int64_t       row_high, 
    const int64_t       src1_ncols,
    const int64_t       src1_padded_row_size, 
    
    hipStream_t        stream) 
    {

    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type  == GGML_TYPE_F32);

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = row_high - row_low;

    GGML_ASSERT(src0->ne[0] == src1->ne[0] && "src0->ne[0] != src1->ne[0]");
    GGML_ASSERT(ggml_cuda_get_tensor_data_axpy(dst->src[2])!=nullptr  && "missing sparse_idx");

    float * sparse_idx = static_cast<float *>(ggml_cuda_get_tensor_data_axpy(dst->src[2]));
    int32_t * gpu_neu_idx = dst->src[3] != NULL ? static_cast<int32_t *>(ggml_cuda_get_tensor_data_axpy(dst->src[3])) : NULL;

    const int cc = ggml_cuda_info().devices[ggml_cuda_get_device()].cc;
    const enum ggml_prec prec = fast_fp16_available(cc) ? ggml_prec(dst->op_params[0]) : GGML_PREC_F32;

    void * src0_d = nullptr;
    switch (src0->type) {
        case GGML_TYPE_F32: {
            const float * src0_d = (const float *) src0_dd_i;
        } break;
        case GGML_TYPE_F16: {
            const half * src0_d = (const half *) src0_dd_i;
        } break;
        case GGML_TYPE_BF16: {
            const hip_bfloat16 * src0_d = (const hip_bfloat16 *) src0_dd_i;
        } break;
        default:
            GGML_ABORT("unsupported type: %s", ggml_type_name(src0->type));
    }

    mul_mat_axpy_cuda_sparse(src0_d, src1_ddf_i, sparse_idx, gpu_neu_idx, dst_dd_i, ncols, nrows, src1_ncols, prec, stream);

    GGML_UNUSED(ctx);
    GGML_UNUSED(src1);
    GGML_UNUSED(dst);
    GGML_UNUSED(src1_ddq_i);
    GGML_UNUSED(src1_ncols);
    GGML_UNUSED(src1_padded_row_size);
}
